
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

// problem parameters
const double a = 1.;
const double b = 1.;

const int nx = 1024; //number of node points along y
const int ny = 1024; //number of node points along x

//convergence parameters
double tol = 1e-4;
int iter_max = 1000;

double sol_ref(double x, double y)
{
   return sin(M_PI*x)*exp(-M_PI*y);
}

void discretisation(double *x, double *y)
{
  double dx = a/nx; // spatial step along x
  double dy = b/ny; // spatial step along y

  for (int i=0; i<=nx; i++)
  {
      x[i] = i*dx;
      y[i] = i*dy;
  }

}

void boundary(double *T, double *x, double *y)
{
      /*Boundary conditions along the x axis for all processes */
      for ( int i=0; i<=nx; i++)
      {
        T[i*nx] = sol_ref(x[i], y[0]);
        T[i*ny+ny] = sol_ref(x[i], y[ny]);
      }

      /*Boundary conditions along the y axis for all processes */
      for ( int j=0; j<=ny; j++)
      {
        T[j] = sol_ref(x[0], y[j]);
        T[nx*ny+j] = sol_ref(x[nx], y[j]);
      }

}

void laplace2d(double *T, double *Tnew, double *error)
{

   for( int j  = 1; j <= nx-1; j++)
   {
     for( int i  = 1; i <= ny-1; i++)
     {
        Tnew[j*nx + i] = 0.25 * ( T[j*nx + (i+1)] + T[j*nx + (i-1)] + T[(j-1)*nx + i] + T[(j+1)*nx + i] );
        *error = fmax(*error, fabs(Tnew[j*nx + i] - T[j*nx + i]));
     }
   }

   for( int j = 1; j <= nx-1; j++)
   {
     for( int i  = 1; i <= ny-1; i++)
     {
        T[j*nx + i] = Tnew[j*nx + i];
     }
   }

}

int main(int argc,char **argv)
{
  double tol = 1e-4;
  int iter = 0;
  double error;
  
  double *T    = (double*) malloc(sizeof(double) * (nx+1) * (ny+1));
  double *Tnew = (double*) malloc(sizeof(double) * (nx+1) * (ny+1));
   
  double *x =  (double*) malloc(nx * sizeof(double));
  double *y =  (double*) malloc(ny * sizeof(double));
  
  if(!x || !y || !T || !Tnew ) return 0;
 
  discretisation(x, y);

  boundary(T, x, y);
    
  while (iter  < iter_max )
  { 
    error = 0.0;
    
    laplace2d(T, Tnew, &error);
    
    if (iter % 100 == 0 ) printf("%d, %0.6f\n", iter, error);

    if (error < tol) break;

    iter++;
  }

  free(T);
  free(Tnew);
  free(x);
  free(y);
  
  return 0;
}
