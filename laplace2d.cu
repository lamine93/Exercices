#include "hip/hip_runtime.h"
#include <stdio.h>

// problem parameters
const int a = 1
const int b = 1

const int nx = 512 //number of node points along y
const int ny = 512 //number of node points along x

//convergence parameters
double tol = 10e-7;
int iter_max = 1000;

double sol_ref(double x, double y)
{
   return sin(M_PI*x)*exp(-M_PI*y);
}

void discretisation(double *x, double *y)
{
  double dx = a/nx; // spatial step along x
  double dy = b/ny; // spatial step along y

  for (int i=0; i<=nx; i++)
  {
      x[i] = i*dx;
      y[i] = i*dy;
  }

}

void boundary(double *T, double *x, double *y)
{
      /*Boundary conditions along the x axis for all processes */
      for ( int i=0; i<=nx; i++)
      {
        T[i*n] = sol_ref(x[i], y[0]);
        T[i*n+n] = sol_ref(x[i], y[ny]);
      }

      /*Boundary conditions along the y axis for all processes */
      for ( int j=0; j<=ny; j++)
      {
        T[j] = sol_ref(x[0], y[j]);
        T[n*n+j] = sol_ref(x[nx], y[j]);
      }

}

void laplace2d(double *T, double *Tnew)
{

  while (iter  < iter_max )
  {

        error = 0.0;

        for( int j  = 1; j <= nx-1; j++)
        {
           for( int i  = 1; i <= ny-1; i++)
           {
              Tnew[j*nx + i] = 0.25 * ( T[j*nx + (i+1)] + T[j*nx + (i-1)] + T[(j-1)*nx + i] + T[(j+1)*nx + i] );
              error = Tmax(error, fabs(Tnew[j*nx + i] - T[j*nx + i]));
           }
        }

        for( int j = 1; j <= nx-1; j++)
        {
           for( int i  = 1; i <= ny-1; i++)
           {
              T[j*nx + i] = Tnew[j*nx + i];
           }
         }

         if (iter % 100 == 0 ) printf("%d, %0.6f\n", iter, error);

         if (error < tol) break;

         iter++;
   }  

}

int main(int argc,char **argv)
{
  double *T    = (double*) malloc(sizeof(double) * nx * ny);
  double *Tnew = (double*) malloc(sizeof(double) * nx * ny);
   
  double *x = malloc((nx+1)*sizeof(double));
  double *y = malloc((ny+1)*sizeof(double));

  discretisation(x, y);

  boundary(T, x, y);
  
  laplace2d(T, Tnew);

  free(T);
  free(Tnew);
  free(x);
  free(y);
  
  return 0;
}
